// vector_add_2d.cu
#include <iostream>
#include <hip/hip_runtime.h>

__device__ float square(float x) {
    return x * x;
}

__global__ void vectorAdd2D(const float* A, const float* B, float* C, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int i = y * width + x;

    if (x < width && y < height) {
        C[i] = square(A[i]) + B[i];
    }
}

int main() {
    int width = 1024;
    int height = 1024;
    int N = width * height;
    size_t size = N * sizeof(float);

    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    for (int i = 0; i < N; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(2 * i);
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((width + 15)/16, (height + 15)/16);
    vectorAdd2D<<<blocks, threads>>>(d_A, d_B, d_C, width, height);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    std::cout << "C[0] = " << h_C[0] << std::endl;

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);
    return 0;
}
