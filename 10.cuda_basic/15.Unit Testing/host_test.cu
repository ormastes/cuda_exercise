#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <vector>

__global__ void simpleKernel(int* result, int value) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    result[idx] = value + idx;
}

__global__ void vectorAddKernel(const float* a, const float* b, float* c, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

TEST(HostLevel, DeviceDetection) {
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);

    EXPECT_EQ(error, hipSuccess);
    EXPECT_GT(deviceCount, 0) << "No CUDA devices found";

    if (deviceCount > 0) {
        hipDeviceProp_t prop;
        error = hipGetDeviceProperties(&prop, 0);
        EXPECT_EQ(error, hipSuccess);
        EXPECT_GT(prop.major, 0);

        std::cout << "Device 0: " << prop.name << std::endl;
        std::cout << "Compute capability: " << prop.major << "." << prop.minor << std::endl;
    }
}

TEST(HostLevel, MemoryOperations) {
    const size_t size = 1024 * sizeof(float);
    float* d_data = nullptr;

    EXPECT_EQ(hipMalloc(&d_data, size), hipSuccess);
    EXPECT_NE(d_data, nullptr);

    EXPECT_EQ(hipMemset(d_data, 0, size), hipSuccess);

    float* h_data = new float[1024];
    for (int i = 0; i < 1024; i++) {
        h_data[i] = static_cast<float>(i);
    }

    EXPECT_EQ(hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice), hipSuccess);

    float* h_result = new float[1024];
    EXPECT_EQ(hipMemcpy(h_result, d_data, size, hipMemcpyDeviceToHost), hipSuccess);

    for (int i = 0; i < 1024; i++) {
        EXPECT_FLOAT_EQ(h_result[i], h_data[i]);
    }

    delete[] h_data;
    delete[] h_result;

    EXPECT_EQ(hipFree(d_data), hipSuccess);
}

TEST(HostLevel, KernelLaunch) {
    const int numElements = 256;
    int* d_result;
    std::vector<int> h_result(numElements);

    ASSERT_EQ(hipMalloc(&d_result, numElements * sizeof(int)), hipSuccess);

    simpleKernel<<<1, numElements>>>(d_result, 100);

    ASSERT_EQ(hipGetLastError(), hipSuccess) << "Kernel launch failed";
    ASSERT_EQ(hipDeviceSynchronize(), hipSuccess) << "Kernel execution failed";

    ASSERT_EQ(hipMemcpy(h_result.data(), d_result, numElements * sizeof(int),
                         hipMemcpyDeviceToHost), hipSuccess);

    for (int i = 0; i < numElements; i++) {
        EXPECT_EQ(h_result[i], 100 + i) << "Mismatch at index " << i;
    }

    hipFree(d_result);
}

TEST(HostLevel, VectorAddition) {
    const int n = 10000;
    const size_t size = n * sizeof(float);

    std::vector<float> h_a(n), h_b(n), h_c(n);
    for (int i = 0; i < n; i++) {
        h_a[i] = static_cast<float>(i);
        h_b[i] = static_cast<float>(i * 2);
    }

    float *d_a, *d_b, *d_c;
    ASSERT_EQ(hipMalloc(&d_a, size), hipSuccess);
    ASSERT_EQ(hipMalloc(&d_b, size), hipSuccess);
    ASSERT_EQ(hipMalloc(&d_c, size), hipSuccess);

    ASSERT_EQ(hipMemcpy(d_a, h_a.data(), size, hipMemcpyHostToDevice), hipSuccess);
    ASSERT_EQ(hipMemcpy(d_b, h_b.data(), size, hipMemcpyHostToDevice), hipSuccess);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    vectorAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    ASSERT_EQ(hipGetLastError(), hipSuccess);
    ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);

    ASSERT_EQ(hipMemcpy(h_c.data(), d_c, size, hipMemcpyDeviceToHost), hipSuccess);

    for (int i = 0; i < n; i++) {
        EXPECT_FLOAT_EQ(h_c[i], h_a[i] + h_b[i]) << "Mismatch at index " << i;
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

class CudaMemoryRAII {
    void* ptr = nullptr;
    size_t size = 0;
public:
    CudaMemoryRAII(size_t s) : size(s) {
        hipMalloc(&ptr, size);
    }
    ~CudaMemoryRAII() {
        if (ptr) hipFree(ptr);
    }
    void* get() { return ptr; }
    operator bool() const { return ptr != nullptr; }
};

TEST(HostLevel, RAIIMemoryPattern) {
    const size_t size = 1024 * sizeof(float);

    {
        CudaMemoryRAII mem(size);
        ASSERT_TRUE(mem);
        ASSERT_NE(mem.get(), nullptr);

        EXPECT_EQ(hipMemset(mem.get(), 0, size), hipSuccess);
    }

    hipError_t error = hipGetLastError();
    EXPECT_EQ(error, hipSuccess);
}

TEST(HostLevel, ErrorHandling) {
    void* ptr = nullptr;

    hipError_t error = hipMalloc(&ptr, SIZE_MAX);
    EXPECT_NE(error, hipSuccess);
    EXPECT_EQ(ptr, nullptr);

    hipGetLastError();

    simpleKernel<<<0, 256>>>(nullptr, 0);
    error = hipGetLastError();
    EXPECT_NE(error, hipSuccess);
}