#include "hip/hip_runtime.h"
// test_memory_api.cu - Unit tests for CUDA Memory API operations
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <memory>
#include <vector>
#include <chrono>
#include <cmath>

// Test fixture for memory API tests
class CudaMemoryTest : public ::testing::Test {
protected:
    void SetUp() override {
        // Ensure CUDA is initialized
        hipError_t err = hipSetDevice(0);
        if (err != hipSuccess) {
            GTEST_SKIP() << "CUDA initialization failed: " << hipGetErrorString(err);
        }
    }

    void TearDown() override {
        // Clean up any remaining allocations
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA error during test: " << hipGetErrorString(err) << std::endl;
        }
        hipDeviceReset();
    }
};

// Simple kernel for testing
__global__ void test_kernel(float* data, int N, float value) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        data[tid] = data[tid] * value;
    }
}

// Test basic allocation and deallocation
TEST_F(CudaMemoryTest, BasicAllocation) {
    const int N = 1024;
    size_t bytes = N * sizeof(float);

    float* d_data = nullptr;
    ASSERT_EQ(hipMalloc(&d_data, bytes), hipSuccess);
    ASSERT_NE(d_data, nullptr);

    // Verify memory is accessible
    ASSERT_EQ(hipMemset(d_data, 0, bytes), hipSuccess);

    ASSERT_EQ(hipFree(d_data), hipSuccess);
}

// Test memory copy operations
TEST_F(CudaMemoryTest, MemoryCopy) {
    const int N = 1024;
    size_t bytes = N * sizeof(float);

    // Allocate host and device memory
    std::unique_ptr<float[]> h_data(new float[N]);
    float* d_data = nullptr;

    ASSERT_EQ(hipMalloc(&d_data, bytes), hipSuccess);

    // Initialize host data
    for (int i = 0; i < N; i++) {
        h_data[i] = (float)i;
    }

    // Copy to device
    ASSERT_EQ(hipMemcpy(d_data, h_data.get(), bytes, hipMemcpyHostToDevice), hipSuccess);

    // Launch kernel
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
    test_kernel<<<gridSize, blockSize>>>(d_data, N, 2.0f);
    ASSERT_EQ(hipGetLastError(), hipSuccess);
    ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);

    // Copy back
    std::unique_ptr<float[]> h_result(new float[N]);
    ASSERT_EQ(hipMemcpy(h_result.get(), d_data, bytes, hipMemcpyDeviceToHost), hipSuccess);

    // Verify
    for (int i = 0; i < N; i++) {
        EXPECT_FLOAT_EQ(h_result[i], h_data[i] * 2.0f);
    }

    hipFree(d_data);
}

// Test pinned memory
TEST_F(CudaMemoryTest, PinnedMemory) {
    const int N = 1024;
    size_t bytes = N * sizeof(float);

    float* h_pinned = nullptr;
    ASSERT_EQ(hipHostAlloc(&h_pinned, bytes, hipHostMallocDefault), hipSuccess);
    ASSERT_NE(h_pinned, nullptr);

    // Initialize
    for (int i = 0; i < N; i++) {
        h_pinned[i] = (float)i;
    }

    float* d_data = nullptr;
    ASSERT_EQ(hipMalloc(&d_data, bytes), hipSuccess);

    // Transfer should work
    ASSERT_EQ(hipMemcpy(d_data, h_pinned, bytes, hipMemcpyHostToDevice), hipSuccess);
    ASSERT_EQ(hipMemcpy(h_pinned, d_data, bytes, hipMemcpyDeviceToHost), hipSuccess);

    // Verify data integrity
    for (int i = 0; i < N; i++) {
        EXPECT_FLOAT_EQ(h_pinned[i], (float)i);
    }

    ASSERT_EQ(hipHostFree(h_pinned), hipSuccess);
    ASSERT_EQ(hipFree(d_data), hipSuccess);
}

// Test unified memory
TEST_F(CudaMemoryTest, UnifiedMemory) {
    const int N = 1024;
    size_t bytes = N * sizeof(float);

    float* unified_data = nullptr;
    ASSERT_EQ(hipMallocManaged(&unified_data, bytes), hipSuccess);
    ASSERT_NE(unified_data, nullptr);

    // Initialize on host
    for (int i = 0; i < N; i++) {
        unified_data[i] = (float)i;
    }

    // Launch kernel
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
    test_kernel<<<gridSize, blockSize>>>(unified_data, N, 3.0f);
    ASSERT_EQ(hipGetLastError(), hipSuccess);
    ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);

    // Verify on host
    for (int i = 0; i < N; i++) {
        EXPECT_FLOAT_EQ(unified_data[i], (float)i * 3.0f);
    }

    ASSERT_EQ(hipFree(unified_data), hipSuccess);
}

// Test zero-copy memory
TEST_F(CudaMemoryTest, ZeroCopyMemory) {
    // Check if device supports mapped memory
    hipDeviceProp_t prop;
    ASSERT_EQ(hipGetDeviceProperties(&prop, 0), hipSuccess);

    if (!prop.canMapHostMemory) {
        GTEST_SKIP() << "Device does not support mapped memory";
    }

    const int N = 1024;
    size_t bytes = N * sizeof(float);

    // Enable mapped memory
    ASSERT_EQ(hipSetDeviceFlags(hipDeviceMapHost), hipSuccess);

    float* h_data = nullptr;
    ASSERT_EQ(hipHostAlloc(&h_data, bytes, hipHostMallocMapped), hipSuccess);
    ASSERT_NE(h_data, nullptr);

    // Get device pointer
    float* d_data = nullptr;
    ASSERT_EQ(hipHostGetDevicePointer(&d_data, h_data, 0), hipSuccess);
    ASSERT_NE(d_data, nullptr);

    // Initialize on host
    for (int i = 0; i < N; i++) {
        h_data[i] = (float)i;
    }

    // Launch kernel with device pointer
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
    test_kernel<<<gridSize, blockSize>>>(d_data, N, 4.0f);
    ASSERT_EQ(hipGetLastError(), hipSuccess);
    ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);

    // Verify on host
    for (int i = 0; i < N; i++) {
        EXPECT_FLOAT_EQ(h_data[i], (float)i * 4.0f);
    }

    ASSERT_EQ(hipHostFree(h_data), hipSuccess);
}

// Test memory info query
TEST_F(CudaMemoryTest, MemoryInfo) {
    size_t free_mem, total_mem;
    ASSERT_EQ(hipMemGetInfo(&free_mem, &total_mem), hipSuccess);

    EXPECT_GT(free_mem, 0);
    EXPECT_GT(total_mem, 0);
    EXPECT_LE(free_mem, total_mem);

    // Allocate some memory and check again
    const size_t alloc_size = 100 * 1024 * 1024; // 100 MB
    float* d_data = nullptr;
    ASSERT_EQ(hipMalloc(&d_data, alloc_size), hipSuccess);

    size_t free_mem_after, total_mem_after;
    ASSERT_EQ(hipMemGetInfo(&free_mem_after, &total_mem_after), hipSuccess);

    // Free memory should have decreased
    EXPECT_LT(free_mem_after, free_mem);
    EXPECT_EQ(total_mem_after, total_mem);

    ASSERT_EQ(hipFree(d_data), hipSuccess);
}

// Test memset operations
TEST_F(CudaMemoryTest, Memset) {
    const int N = 1024;
    size_t bytes = N * sizeof(float);

    float* d_data = nullptr;
    ASSERT_EQ(hipMalloc(&d_data, bytes), hipSuccess);

    // Set to zero
    ASSERT_EQ(hipMemset(d_data, 0, bytes), hipSuccess);

    // Verify
    std::unique_ptr<float[]> h_data(new float[N]);
    ASSERT_EQ(hipMemcpy(h_data.get(), d_data, bytes, hipMemcpyDeviceToHost), hipSuccess);

    for (int i = 0; i < N; i++) {
        EXPECT_FLOAT_EQ(h_data[i], 0.0f);
    }

    ASSERT_EQ(hipFree(d_data), hipSuccess);
}

// Test 2D memory operations
TEST_F(CudaMemoryTest, Memory2D) {
    const int width = 64;
    const int height = 32;
    size_t pitch;

    float* d_data = nullptr;
    ASSERT_EQ(hipMallocPitch(&d_data, &pitch, width * sizeof(float), height), hipSuccess);
    ASSERT_NE(d_data, nullptr);
    EXPECT_GE(pitch, width * sizeof(float));

    // Prepare host data
    std::vector<float> h_data(width * height);
    for (int i = 0; i < width * height; i++) {
        h_data[i] = (float)i;
    }

    // Copy to device
    ASSERT_EQ(hipMemcpy2D(d_data, pitch,
                           h_data.data(), width * sizeof(float),
                           width * sizeof(float), height,
                           hipMemcpyHostToDevice), hipSuccess);

    // Copy back
    std::vector<float> h_result(width * height);
    ASSERT_EQ(hipMemcpy2D(h_result.data(), width * sizeof(float),
                           d_data, pitch,
                           width * sizeof(float), height,
                           hipMemcpyDeviceToHost), hipSuccess);

    // Verify
    for (int i = 0; i < width * height; i++) {
        EXPECT_FLOAT_EQ(h_result[i], h_data[i]);
    }

    ASSERT_EQ(hipFree(d_data), hipSuccess);
}

// Test async memory operations
TEST_F(CudaMemoryTest, AsyncOperations) {
    const int N = 1024;
    size_t bytes = N * sizeof(float);

    // Create stream
    hipStream_t stream;
    ASSERT_EQ(hipStreamCreate(&stream), hipSuccess);

    // Allocate memory
    std::unique_ptr<float[]> h_data(new float[N]);
    float* d_data = nullptr;
    ASSERT_EQ(hipMalloc(&d_data, bytes), hipSuccess);

    // Initialize
    for (int i = 0; i < N; i++) {
        h_data[i] = (float)i;
    }

    // Async copy
    ASSERT_EQ(hipMemcpyAsync(d_data, h_data.get(), bytes,
                              hipMemcpyHostToDevice, stream), hipSuccess);

    // Launch kernel on same stream
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
    test_kernel<<<gridSize, blockSize, 0, stream>>>(d_data, N, 5.0f);
    ASSERT_EQ(hipGetLastError(), hipSuccess);

    // Async copy back
    std::unique_ptr<float[]> h_result(new float[N]);
    ASSERT_EQ(hipMemcpyAsync(h_result.get(), d_data, bytes,
                              hipMemcpyDeviceToHost, stream), hipSuccess);

    // Wait for stream
    ASSERT_EQ(hipStreamSynchronize(stream), hipSuccess);

    // Verify
    for (int i = 0; i < N; i++) {
        EXPECT_FLOAT_EQ(h_result[i], h_data[i] * 5.0f);
    }

    ASSERT_EQ(hipStreamDestroy(stream), hipSuccess);
    ASSERT_EQ(hipFree(d_data), hipSuccess);
}

// Test memory bandwidth
TEST_F(CudaMemoryTest, MemoryBandwidth) {
    const int N = 10 * 1024 * 1024; // 10M elements
    size_t bytes = N * sizeof(float);

    // Skip if not enough memory
    size_t free_mem, total_mem;
    ASSERT_EQ(hipMemGetInfo(&free_mem, &total_mem), hipSuccess);
    if (free_mem < bytes * 2) {
        GTEST_SKIP() << "Not enough GPU memory for bandwidth test";
    }

    std::unique_ptr<float[]> h_data(new float[N]);
    float* d_data = nullptr;
    ASSERT_EQ(hipMalloc(&d_data, bytes), hipSuccess);

    // Initialize
    for (int i = 0; i < N; i++) {
        h_data[i] = (float)i;
    }

    // Measure H2D bandwidth
    auto start = std::chrono::high_resolution_clock::now();
    ASSERT_EQ(hipMemcpy(d_data, h_data.get(), bytes, hipMemcpyHostToDevice), hipSuccess);
    auto end = std::chrono::high_resolution_clock::now();

    float h2d_time = std::chrono::duration<float, std::milli>(end - start).count();
    float h2d_bandwidth = (bytes / (1024.0f * 1024.0f * 1024.0f)) / (h2d_time / 1000.0f);

    // Measure D2H bandwidth
    start = std::chrono::high_resolution_clock::now();
    ASSERT_EQ(hipMemcpy(h_data.get(), d_data, bytes, hipMemcpyDeviceToHost), hipSuccess);
    end = std::chrono::high_resolution_clock::now();

    float d2h_time = std::chrono::duration<float, std::milli>(end - start).count();
    float d2h_bandwidth = (bytes / (1024.0f * 1024.0f * 1024.0f)) / (d2h_time / 1000.0f);

    std::cout << "\nMemory Bandwidth Test:" << std::endl;
    std::cout << "  H2D: " << h2d_bandwidth << " GB/s" << std::endl;
    std::cout << "  D2H: " << d2h_bandwidth << " GB/s" << std::endl;

    // Basic sanity check - bandwidth should be positive
    EXPECT_GT(h2d_bandwidth, 0.0f);
    EXPECT_GT(d2h_bandwidth, 0.0f);

    ASSERT_EQ(hipFree(d_data), hipSuccess);
}

// Test pinned vs pageable memory performance
TEST_F(CudaMemoryTest, PinnedVsPageable) {
    const int N = 10 * 1024 * 1024; // 10M elements
    size_t bytes = N * sizeof(float);

    // Allocate pageable memory
    std::unique_ptr<float[]> h_pageable(new float[N]);

    // Allocate pinned memory
    float* h_pinned = nullptr;
    ASSERT_EQ(hipHostAlloc(&h_pinned, bytes, hipHostMallocDefault), hipSuccess);

    // Allocate device memory
    float* d_data = nullptr;
    ASSERT_EQ(hipMalloc(&d_data, bytes), hipSuccess);

    // Initialize both
    for (int i = 0; i < N; i++) {
        h_pageable[i] = (float)i;
        h_pinned[i] = (float)i;
    }

    // Measure pageable transfer
    auto start = std::chrono::high_resolution_clock::now();
    ASSERT_EQ(hipMemcpy(d_data, h_pageable.get(), bytes, hipMemcpyHostToDevice), hipSuccess);
    auto end = std::chrono::high_resolution_clock::now();
    float pageable_time = std::chrono::duration<float, std::milli>(end - start).count();

    // Measure pinned transfer
    start = std::chrono::high_resolution_clock::now();
    ASSERT_EQ(hipMemcpy(d_data, h_pinned, bytes, hipMemcpyHostToDevice), hipSuccess);
    end = std::chrono::high_resolution_clock::now();
    float pinned_time = std::chrono::duration<float, std::milli>(end - start).count();

    std::cout << "\nPinned vs Pageable Memory:" << std::endl;
    std::cout << "  Pageable: " << pageable_time << " ms" << std::endl;
    std::cout << "  Pinned: " << pinned_time << " ms" << std::endl;
    std::cout << "  Speedup: " << pageable_time / pinned_time << "x" << std::endl;

    // Pinned should generally be faster or at least not slower
    EXPECT_LE(pinned_time, pageable_time * 1.1f); // Allow 10% margin

    ASSERT_EQ(hipHostFree(h_pinned), hipSuccess);
    ASSERT_EQ(hipFree(d_data), hipSuccess);
}