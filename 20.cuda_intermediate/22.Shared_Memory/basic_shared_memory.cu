#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>

#define BLOCK_SIZE 256
#define ARRAY_SIZE 1048576

__global__ void reverseArrayGlobal(float* d_in, float* d_out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        d_out[idx] = d_in[n - 1 - idx];
    }
}

__global__ void reverseArrayShared(float* d_in, float* d_out, int n) {
    __shared__ float sharedData[BLOCK_SIZE];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;

    if (idx < n) {
        int reverseIdx = n - 1 - idx;
        int reverseBlockIdx = reverseIdx / blockDim.x;
        int reverseTid = reverseIdx % blockDim.x;

        if (reverseBlockIdx == blockIdx.x) {
            sharedData[tid] = d_in[idx];
            __syncthreads();
            d_out[idx] = sharedData[blockDim.x - 1 - tid];
        } else {
            d_out[idx] = d_in[reverseIdx];
        }
    }
}

__global__ void sumReductionGlobal(float* d_in, float* d_partial, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int blockStart = blockIdx.x * blockDim.x;

    float sum = 0.0f;
    if (idx < n) {
        sum = d_in[idx];
    }

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        __syncthreads();
        if (tid < stride && idx + stride < n) {
            sum += d_in[blockStart + tid + stride];
        }
        if (tid < stride) {
            d_in[blockStart + tid] = sum;
        }
    }

    if (tid == 0) {
        d_partial[blockIdx.x] = sum;
    }
}

__global__ void sumReductionShared(float* d_in, float* d_partial, int n) {
    __shared__ float sharedData[BLOCK_SIZE];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    sharedData[tid] = (idx < n) ? d_in[idx] : 0.0f;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sharedData[tid] += sharedData[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        d_partial[blockIdx.x] = sharedData[0];
    }
}

void checkCudaError(hipError_t error, const char* function) {
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error in %s: %s\n", function, hipGetErrorString(error));
        exit(1);
    }
}

int main() {
    float *h_in, *h_out;
    float *d_in, *d_out, *d_partial;

    size_t size = ARRAY_SIZE * sizeof(float);
    int numBlocks = (ARRAY_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE;
    size_t partialSize = numBlocks * sizeof(float);

    h_in = (float*)malloc(size);
    h_out = (float*)malloc(size);

    for (int i = 0; i < ARRAY_SIZE; i++) {
        h_in[i] = (float)(i + 1);
    }

    checkCudaError(hipMalloc(&d_in, size), "hipMalloc d_in");
    checkCudaError(hipMalloc(&d_out, size), "hipMalloc d_out");
    checkCudaError(hipMalloc(&d_partial, partialSize), "hipMalloc d_partial");

    checkCudaError(hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice), "hipMemcpy to device");

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;

    printf("Array Reversal Comparison (Array size: %d):\n", ARRAY_SIZE);
    printf("=============================================\n");

    hipEventRecord(start);
    reverseArrayGlobal<<<numBlocks, BLOCK_SIZE>>>(d_in, d_out, ARRAY_SIZE);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Global memory version: %.3f ms\n", milliseconds);

    hipMemcpy(d_out, h_in, size, hipMemcpyHostToDevice);

    hipEventRecord(start);
    reverseArrayShared<<<numBlocks, BLOCK_SIZE>>>(d_in, d_out, ARRAY_SIZE);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Shared memory version: %.3f ms\n", milliseconds);

    checkCudaError(hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost), "hipMemcpy to host");

    bool correct = true;
    for (int i = 0; i < ARRAY_SIZE && i < 10; i++) {
        if (h_out[i] != h_in[ARRAY_SIZE - 1 - i]) {
            correct = false;
            printf("Error at index %d: expected %.0f, got %.0f\n",
                   i, h_in[ARRAY_SIZE - 1 - i], h_out[i]);
        }
    }
    if (correct) {
        printf("Array reversal: PASSED\n");
    }

    printf("\nSum Reduction Comparison:\n");
    printf("=============================================\n");

    checkCudaError(hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice), "hipMemcpy to device");

    float* d_in_copy;
    checkCudaError(hipMalloc(&d_in_copy, size), "hipMalloc d_in_copy");
    checkCudaError(hipMemcpy(d_in_copy, d_in, size, hipMemcpyDeviceToDevice), "hipMemcpy d_in_copy");

    hipEventRecord(start);
    sumReductionGlobal<<<numBlocks, BLOCK_SIZE>>>(d_in_copy, d_partial, ARRAY_SIZE);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Global memory reduction: %.3f ms\n", milliseconds);

    checkCudaError(hipMemcpy(d_in_copy, d_in, size, hipMemcpyDeviceToDevice), "hipMemcpy restore");

    hipEventRecord(start);
    sumReductionShared<<<numBlocks, BLOCK_SIZE>>>(d_in_copy, d_partial, ARRAY_SIZE);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Shared memory reduction: %.3f ms\n", milliseconds);

    float* h_partial = (float*)malloc(partialSize);
    checkCudaError(hipMemcpy(h_partial, d_partial, partialSize, hipMemcpyDeviceToHost), "hipMemcpy partial");

    float gpuSum = 0.0f;
    for (int i = 0; i < numBlocks; i++) {
        gpuSum += h_partial[i];
    }

    double expectedSum = (double)ARRAY_SIZE * ((double)ARRAY_SIZE + 1.0) / 2.0;
    printf("\nSum verification:\n");
    printf("Expected sum: %.0f\n", expectedSum);
    printf("GPU sum: %.0f\n", gpuSum);
    double relativeError = fabs(gpuSum - expectedSum) / expectedSum;
    if (relativeError < 0.001) {
        printf("Sum reduction: PASSED\n");
    } else {
        printf("Sum reduction: FAILED\n");
    }

    free(h_in);
    free(h_out);
    free(h_partial);
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_in_copy);
    hipFree(d_partial);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}