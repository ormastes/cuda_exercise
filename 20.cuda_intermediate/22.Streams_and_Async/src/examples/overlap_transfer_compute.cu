#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>

#define CHUNK_SIZE (1<<18)
#define NCHUNKS 16
#define NSTREAMS 2

__global__ void process_data(float* data, int size, float factor) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float val = data[idx];
        for (int i = 0; i < 100; i++) {
            val = sinf(val) * factor + cosf(val);
        }
        data[idx] = val;
    }
}

void check_cuda_error(hipError_t error, const char* function) {
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error in %s: %s\n", function, hipGetErrorString(error));
        exit(1);
    }
}

void init_data(float* data, int size) {
    for (int i = 0; i < size; i++) {
        data[i] = (float)(i % 1000) / 1000.0f;
    }
}

int main() {
    printf("Overlapping Transfer and Compute Example\n");
    printf("=========================================\n");
    printf("Total data size: %d chunks of %d elements\n", NCHUNKS, CHUNK_SIZE);
    printf("Number of streams: %d\n\n", NSTREAMS);

    size_t chunkBytes = CHUNK_SIZE * sizeof(float);
    size_t totalSize = NCHUNKS * CHUNK_SIZE;
    size_t totalBytes = totalSize * sizeof(float);

    float *h_data_in, *h_data_out;
    float *d_data[NSTREAMS];

    hipHostMalloc(&h_data_in, totalBytes, hipHostMallocDefault);
    hipHostMalloc(&h_data_out, totalBytes, hipHostMallocDefault);

    for (int i = 0; i < NSTREAMS; i++) {
        check_cuda_error(hipMalloc(&d_data[i], chunkBytes), "hipMalloc");
    }

    init_data(h_data_in, totalSize);

    hipStream_t streams[NSTREAMS];
    for (int i = 0; i < NSTREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;

    int threadsPerBlock = 256;
    int blocksPerGrid = (CHUNK_SIZE + threadsPerBlock - 1) / threadsPerBlock;

    printf("Performance Comparison:\n");
    printf("-----------------------\n");

    float* d_data_seq;
    check_cuda_error(hipMalloc(&d_data_seq, totalBytes), "hipMalloc sequential");

    hipEventRecord(start);
    hipMemcpy(d_data_seq, h_data_in, totalBytes, hipMemcpyHostToDevice);
    for (int i = 0; i < NCHUNKS; i++) {
        process_data<<<blocksPerGrid, threadsPerBlock>>>(d_data_seq + i * CHUNK_SIZE, CHUNK_SIZE, 2.0f);
    }
    hipMemcpy(h_data_out, d_data_seq, totalBytes, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Sequential (no overlap):     %.3f ms\n", milliseconds);
    float sequentialTime = milliseconds;

    memset(h_data_out, 0, totalBytes);
    hipEventRecord(start);

    for (int i = 0; i < NCHUNKS; i++) {
        int streamId = i % NSTREAMS;
        int offset = i * CHUNK_SIZE;

        hipMemcpyAsync(d_data[streamId], h_data_in + offset, chunkBytes,
                       hipMemcpyHostToDevice, streams[streamId]);

        process_data<<<blocksPerGrid, threadsPerBlock, 0, streams[streamId]>>>(
            d_data[streamId], CHUNK_SIZE, 2.0f);

        hipMemcpyAsync(h_data_out + offset, d_data[streamId], chunkBytes,
                       hipMemcpyDeviceToHost, streams[streamId]);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Overlapped (%d streams):      %.3f ms (%.2fx speedup)\n",
           NSTREAMS, milliseconds, sequentialTime / milliseconds);
    float overlappedTime = milliseconds;

    memset(h_data_out, 0, totalBytes);
    hipEventRecord(start);

    for (int i = 0; i < NCHUNKS; i++) {
        int streamId = i % NSTREAMS;
        int offset = i * CHUNK_SIZE;
        hipMemcpyAsync(d_data[streamId], h_data_in + offset, chunkBytes,
                       hipMemcpyHostToDevice, streams[streamId]);
    }

    for (int i = 0; i < NCHUNKS; i++) {
        int streamId = i % NSTREAMS;
        process_data<<<blocksPerGrid, threadsPerBlock, 0, streams[streamId]>>>(
            d_data[streamId], CHUNK_SIZE, 2.0f);
    }

    for (int i = 0; i < NCHUNKS; i++) {
        int streamId = i % NSTREAMS;
        int offset = i * CHUNK_SIZE;
        hipMemcpyAsync(h_data_out + offset, d_data[streamId], chunkBytes,
                       hipMemcpyDeviceToHost, streams[streamId]);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Batched operations:          %.3f ms (%.2fx speedup)\n",
           milliseconds, sequentialTime / milliseconds);

    printf("\nTransfer and Compute Analysis:\n");
    printf("-------------------------------\n");

    hipEventRecord(start);
    for (int i = 0; i < NCHUNKS; i++) {
        hipMemcpy(d_data_seq + i * CHUNK_SIZE, h_data_in + i * CHUNK_SIZE,
                  chunkBytes, hipMemcpyHostToDevice);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    float h2dTime = milliseconds;
    printf("H2D Transfer only:           %.3f ms\n", h2dTime);

    hipEventRecord(start);
    for (int i = 0; i < NCHUNKS; i++) {
        process_data<<<blocksPerGrid, threadsPerBlock>>>(
            d_data_seq + i * CHUNK_SIZE, CHUNK_SIZE, 2.0f);
    }
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    float computeTime = milliseconds;
    printf("Compute only:                %.3f ms\n", computeTime);

    hipEventRecord(start);
    for (int i = 0; i < NCHUNKS; i++) {
        hipMemcpy(h_data_out + i * CHUNK_SIZE, d_data_seq + i * CHUNK_SIZE,
                  chunkBytes, hipMemcpyDeviceToHost);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    float d2hTime = milliseconds;
    printf("D2H Transfer only:           %.3f ms\n", d2hTime);

    float theoreticalMin = fmax(fmax(h2dTime, computeTime), d2hTime);
    printf("\nTheoretical minimum time:    %.3f ms\n", theoreticalMin);
    printf("Overlap efficiency:          %.1f%%\n",
           (theoreticalMin / overlappedTime) * 100);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("\nDevice Capabilities:\n");
    printf("--------------------\n");
    printf("Device: %s\n", prop.name);
    printf("Concurrent kernels:          %s\n", prop.concurrentKernels ? "Yes" : "No");
    printf("Async engine count:          %d\n", prop.asyncEngineCount);
    printf("Unified addressing:          %s\n",
           prop.unifiedAddressing ? "Yes" : "No");

    for (int i = 0; i < NSTREAMS; i++) {
        hipStreamDestroy(streams[i]);
    }

    hipHostFree(h_data_in);
    hipHostFree(h_data_out);
    for (int i = 0; i < NSTREAMS; i++) {
        hipFree(d_data[i]);
    }
    hipFree(d_data_seq);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}