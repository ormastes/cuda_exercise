#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N (1<<20)
#define NSTREAMS 4

__global__ void vectorAdd(float* a, float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

void checkCudaError(hipError_t error, const char* function) {
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error in %s: %s\n", function, hipGetErrorString(error));
        exit(1);
    }
}

void initData(float* data, int size) {
    for (int i = 0; i < size; i++) {
        data[i] = (float)(rand() % 100) / 100.0f;
    }
}

int main() {
    printf("CUDA Streams Basic Example\n");
    printf("===========================\n");
    printf("Vector size: %d elements\n", N);
    printf("Number of streams: %d\n\n", NSTREAMS);

    size_t size = N * sizeof(float);
    int streamSize = N / NSTREAMS;
    size_t streamBytes = streamSize * sizeof(float);

    float *h_a, *h_b, *h_c, *h_c_ref;
    float *d_a, *d_b, *d_c;

    hipHostMalloc(&h_a, size, hipHostMallocDefault);
    hipHostMalloc(&h_b, size, hipHostMallocDefault);
    hipHostMalloc(&h_c, size, hipHostMallocDefault);
    h_c_ref = (float*)malloc(size);

    checkCudaError(hipMalloc(&d_a, size), "hipMalloc d_a");
    checkCudaError(hipMalloc(&d_b, size), "hipMalloc d_b");
    checkCudaError(hipMalloc(&d_c, size), "hipMalloc d_c");

    initData(h_a, N);
    initData(h_b, N);

    for (int i = 0; i < N; i++) {
        h_c_ref[i] = h_a[i] + h_b[i];
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;

    int threadsPerBlock = 256;
    int blocksPerGrid = (streamSize + threadsPerBlock - 1) / threadsPerBlock;

    printf("Execution Time Comparison:\n");
    printf("--------------------------\n");

    hipEventRecord(start);
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    vectorAdd<<<blocksPerGrid * NSTREAMS, threadsPerBlock>>>(d_a, d_b, d_c, N);
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Sequential execution (no streams): %.3f ms\n", milliseconds);

    bool correct = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_c[i] - h_c_ref[i]) > 1e-5) {
            correct = false;
            break;
        }
    }
    printf("Result verification: %s\n\n", correct ? "PASSED" : "FAILED");

    hipStream_t streams[NSTREAMS];
    for (int i = 0; i < NSTREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    hipEventRecord(start);
    for (int i = 0; i < NSTREAMS; i++) {
        int offset = i * streamSize;
        hipMemcpyAsync(&d_a[offset], &h_a[offset], streamBytes, hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(&d_b[offset], &h_b[offset], streamBytes, hipMemcpyHostToDevice, streams[i]);
        vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, streams[i]>>>(&d_a[offset], &d_b[offset], &d_c[offset], streamSize);
        hipMemcpyAsync(&h_c[offset], &d_c[offset], streamBytes, hipMemcpyDeviceToHost, streams[i]);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Async execution (%d streams):      %.3f ms\n", NSTREAMS, milliseconds);

    correct = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_c[i] - h_c_ref[i]) > 1e-5) {
            correct = false;
            break;
        }
    }
    printf("Result verification: %s\n\n", correct ? "PASSED" : "FAILED");

    memset(h_c, 0, size);
    hipEventRecord(start);

    for (int i = 0; i < NSTREAMS; i++) {
        int offset = i * streamSize;
        hipMemcpyAsync(&d_a[offset], &h_a[offset], streamBytes, hipMemcpyHostToDevice, streams[i]);
    }
    for (int i = 0; i < NSTREAMS; i++) {
        int offset = i * streamSize;
        hipMemcpyAsync(&d_b[offset], &h_b[offset], streamBytes, hipMemcpyHostToDevice, streams[i]);
    }
    for (int i = 0; i < NSTREAMS; i++) {
        int offset = i * streamSize;
        vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, streams[i]>>>(&d_a[offset], &d_b[offset], &d_c[offset], streamSize);
    }
    for (int i = 0; i < NSTREAMS; i++) {
        int offset = i * streamSize;
        hipMemcpyAsync(&h_c[offset], &d_c[offset], streamBytes, hipMemcpyDeviceToHost, streams[i]);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Batched operations (%d streams):   %.3f ms\n", NSTREAMS, milliseconds);

    correct = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_c[i] - h_c_ref[i]) > 1e-5) {
            correct = false;
            break;
        }
    }
    printf("Result verification: %s\n\n", correct ? "PASSED" : "FAILED");

    printf("Stream Properties:\n");
    printf("------------------\n");
    for (int i = 0; i < NSTREAMS; i++) {
        hipStreamQuery(streams[i]);
        printf("Stream %d: Created and functional\n", i);
    }

    for (int i = 0; i < NSTREAMS; i++) {
        hipStreamDestroy(streams[i]);
    }

    free(h_c_ref);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}